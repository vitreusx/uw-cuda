#include "stream.cuh"
#include "utils.cuh"

Stream::Stream() {
    check(hipStreamCreate(&stream));
}

Stream::~Stream() {
    check(hipStreamDestroy(stream));
    stream = 0;
}
