#include "stream.h"
#include "utils.h"

Stream::Stream() {
    check(hipStreamCreate(&stream));
}

Stream::~Stream() {
    check(hipStreamDestroy(stream));
    stream = 0;
}

Stream::operator hipStream_t() {
    return stream;
}
