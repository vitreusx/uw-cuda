#include "utils.h"
#include <stdexcept>
using namespace std;

void check(hipError_t ret) {
    if (ret != hipSuccess) {
        throw runtime_error(hipGetErrorString(ret));
    }
}

void check() {
    check(hipGetLastError());
}
