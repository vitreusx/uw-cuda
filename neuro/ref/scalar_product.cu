#include "hip/hip_runtime.h"
//
//  scalar_product.c
//  
//
//  Created by Witold Rudnicki on 09.03.2020.
//

#include "scalar_product.h"
int ReadCSV(char* CSVfile,float** CNV) {
// We know the size of the data, we define sufficiently large buffers for this case.
//
    FILE* CNVfile;
    float* row;
    const int buf_size=2000000;
    const int max_cols=100000;
    char buffer[buf_size];
    char* tmp;
    //char head[110]; //for debugging only
    int col_count;
    
    row = (float*) malloc(max_cols*sizeof(float));
    printf("Reading %s\n",CSVfile);
    CNVfile = fopen(CSVfile,"r");
    int line_count =0;
    
    int row_count;
    while ((tmp=fgets(buffer, 1999999, CNVfile))!=NULL){
        line_count++;
        if (line_count>1){
            //we skip header line, hence we start from line_count = 2 here.
            row_count = line_count-2;
            //for (int i=0;i<30;i++) head[i]=buffer[i];
            //printf("line %d starts as %30s\n",line_count,head);
            col_count=-1;
            char *col = strtok(buffer, ",");
            while (col) {
                if (col_count >= 0) {
                    row[col_count]=atof(col);
                }
                col = strtok(NULL, ",");
                col_count++;
            }
            //printf("converted entire line of %d columns\n",col_count);
            //printf("%8.3f, %8.3f %8.3f\n",row[0],row[1],row[2]);
            CNV[row_count]= (float*) malloc((col_count+1)*sizeof(float));
            for (int i=0;i<=col_count;i++) CNV[row_count][i]=row[i];
            //printf("row_count %d\n",row_count);
            //printf("&CNV[%d] = %d\n",row_count,CNV[row_count]);
            //for (int i=0;i<10;i++) printf("%f, ",CNV[row_count][i]);
            //printf("\n");
        }
    }
    //printf("%i\n",CNV);
    //printf("%f\n",CNV[0][0]);
    fclose(CNVfile);
    //for (int i=0;i<10;i++) {
    //    for (int j=0;j<10;j++) printf("%f, ",CNV[i][j]);
    //    printf("\n");
    //}
    return(col_count);
}


float scalar(float *x, float* y,int len){
    //printf("Multiplying vectors\n");
    float sum=0.0f;
    for (int i=0;i<len;i++) sum+= x[i]*y[i];
    return(sum);
}

void scalar_gpu(float *x, float* y, int len, float* z, float* sum){
    //printf("Multiplying vectors\n");
    //float* z;
    //z = malloc(sizeof(float)*len);
    *sum =0.0f;
    
    for (int i=0;i<len;i++) z[i] = x[i]*y[i];

    for (int i=0;i<len;i++) *sum += z[i];
    // tu robimy redukcję na GPU
    //return(sum);
}


double scalar_d(float *x, float* y,int len){
    //printf("Multiplying vectors\n");
    double sum=0.0;
    for (int i=0;i<len;i++) sum+= x[i]*y[i];
    return(sum);
}


int main(int argc, char** argv){
    const int ROWS=145;
    //float x[1], y[1];
    float* CNV[ROWS];
    float* normCNV[ROWS];
    //float scl;
    //double scl_d;
    float* SimMat;
    
    int len;
    if (argc==2) {
        len=ReadCSV(argv[1],CNV);
        // ReadCSV returns the length of the vectors,
        // It also fills two-dimensional array CNV.
        // The rows of CNV are allocated in the function.
        //
        // Allocation of the memory for the normalized version of CNV array.
        for (int i=0;i<ROWS;i++) normCNV[i]= (float*) malloc(len*sizeof(float));
        // Normalization
        normalize(CNV,normCNV,ROWS,len);
        //printf("Lengths of normalized vectors:\n");
        //for (int i=0;i<ROWS;i++){
        //    scl = scalar(normCNV[i],normCNV[i],len);
        //    printf("%12.8f ",scl);
        // }
        //printf("\n");
        //normalize_d(CNV,normCNV,ROWS,len);
        //for (int i=0;i<ROWS;i++){
        //    scl_d = scalar_d(normCNV[i],normCNV[i],len);
        //    printf("%12.8lf ",scl_d);
        //}
        //
        // Computation of similarity matrix
        //
        SimMat= (float*) malloc(ROWS*ROWS*sizeof(float));
        //similarity(normCNV,SimMat,ROWS,len);
        similarity_gpu(normCNV,SimMat,ROWS,len);
        return(0);
    }
    else {
        printf("Wrong number of arguments\n");
        printf("Usage: %s filename\nExiting\n",argv[0]);
        exit(1);
    }
}

void normalize(float** sourceMat,float**  destMat, int Size, int Len) {
    int i,j;
    float Sum;
    // Zdefiniuj timery
    time_t start, end;
    // do computations
    printf("function normalize():\n");
    start=clock();
    for (i=0;i<Size;i++){
        Sum=scalar(sourceMat[i],sourceMat[i],Len);
        Sum=sqrt(Sum);
        for (j=0;j<Len;j++) destMat[i][j]=sourceMat[i][j]/Sum;
        //printf("%8.4f ",Sum);
    }
    end=clock();
    printf("computations took %lf s\n\n",1.0*(end-start)/CLOCKS_PER_SEC);
    
}

void normalize_d(float** sourceMat,float**  destMat, int Size, int Len) {
    int i,j;
    double Sum;
    float Sum_f;
    // Zdefiniuj timery
    time_t start, end;
    // do computations
    start=clock();
    for (i=0;i<Size;i++){
        Sum=scalar_d(sourceMat[i],sourceMat[i],Len);
        Sum_f= (float) sqrt(Sum);
        for (j=0;j<Len;j++) destMat[i][j]=sourceMat[i][j]/Sum_f;
        printf("%8.4lf ",Sum);
    }
    end=clock();
    printf("\nComputations took %lf s\n\n",1.0*(end-start)/CLOCKS_PER_SEC);
}


void similarity(float** sourceMat, float* simMat, int Size, int Len){
    int i,j;
    float Sum, Min, Max;
    Sum=Max=0.0f;
    Min=1.0f;
    time_t start, end;
    // do computations
    printf("function similarity()\n:");
    start=clock();
    for (i=0;i<Size;i++) {
        for (j=0;j<Size;j++) {
            simMat[i*Size+j]=scalar(sourceMat[i],sourceMat[j],Len);
            //Sum+=simMat[i*Size+j];
            //if (Min>fabs(simMat[i*Size+j])) Min=fabs(simMat[i*Size+j]);
            //if ( (i!=j) & (Max<simMat[i*Size+j]) ) Max=simMat[i*Size+j];
        }
    }
    end=clock();
    printf("\ncomputations took %lf s\n\n",1.0*(end-start)/CLOCKS_PER_SEC);
    printf("Minimum similarity(%%): %f\n",Min*100);
    printf("Maximum similarity(%%): %f\n",Max*100);
    printf("Average similarity(%%): %f\n",(Sum-Size)/(Size*(Size-1))*100);
}

void similarity_gpu(float** sourceMat, float* simMat, int Size, int Len){
    int i,j, indx;
    float Sum, Min, Max;
    Sum=Max=0.0f;
    Min=1.0f;
    float *gpuMat;
    float *cpuMat;
    
    float *simMat_cpu;
    float *simMat_gpu;
    hipError_t status;
    time_t start, end;
    
    start=clock();
    // Make a local copy of the CPU matrix
    // It will be easier to copy it in a single operation to GPU
    status = hipHostMalloc((void**) &cpuMat, sizeof(float) * Size*Len);
    if (status != hipSuccess) { cout << hipGetErrorString(status) << endl; };
    // copy vectors from sourceMat to cpuMat
    indx=0;
    for (i =0;i<Size;i++) {
        for (j=0;j<Len;j++) {
            cpuMat[indx]=sourceMat[i][j];
            indx++;
        }
    }
    end=clock();
    printf("\n Memory allocation and copying  %lf s\n\n",1.0*(end-start)/CLOCKS_PER_SEC);
    // Allocat a local CPU similarity matrix
    status = hipHostMalloc((void**) &simMat_cpu, sizeof(float) * Size*Size);
    if (status != hipSuccess) { cout << hipGetErrorString(status) << endl; };

    // Allocate GPU tables
    status = hipMalloc((void**) &gpuMat, sizeof(float) * Size*Len);
    if (status != hipSuccess) { cout << hipGetErrorString(status) << endl; };

    status = hipMalloc((void**) &simMat_gpu, sizeof(float) * Size*Size);
    if (status != hipSuccess) { cout << hipGetErrorString(status) << endl; };

    // Copy CPU Table to GPU
    status = hipMemcpy(gpuMat, cpuMat, sizeof(float) * Size*Len, hipMemcpyHostToDevice);
    if (status != hipSuccess){ cout << hipGetErrorString(status) << endl; };
    
    
    // do computations
    printf("function similarity()\n:");
    start=clock();
    for (i=0;i<Size;i++) {
        for (j=0;j<Size;j++) {
            simMat[i*Size+j]=scalar(sourceMat[i],sourceMat[j],Len);
        }
    }
    end=clock();
    printf("\n CPU computations took %lf s\n\n",1.0*(end-start)/CLOCKS_PER_SEC);
    
    Min =1.0;
    Max =0.0;
    float r, r2;
    Sum = 0;

    for (i=0;i<Size;i++) {
        for (j=i+1;j<Size;j++) {
            r = simMat[i*Size+j];
            r2 = r*r;
            if (r2 < Min) Min=r2;
            if (r2>Max) Max = r2;
            Sum += r2;
        }
    }
    
    printf("Minimum similarity(%%): %f\n",Min*100);
    printf("Maximum similarity(%%): %f\n",Max*100);
    printf("Average similarity(%%): %f\n",Sum/(Size*(Size-1))/2*100);
    #define KERNEL 1
    if (KERNEL == 1) {
        // kernel 1
        //int limit =29;
        int limit =Size;
        start=clock();
        for (i=0;i<limit;i++) {
            for (j=0;j<limit;j++) {
                scalar_1<<< 1, 256, 0>>>(gpuMat , i, j , Len, Size, simMat_gpu );
                //scalar_2<<< 1, BlockSize, 0>>>(gpuMat , i, j , Len, Size, simMat_gpu );
                //simMat[i*Size+j]=scalar(sourceMat[i],sourceMat[j],Len);            
            }
        }
        status = hipMemcpy(simMat_cpu, simMat_gpu, sizeof(int)* Size*Size, hipMemcpyDeviceToHost);
        if (status != hipSuccess){ cout << hipGetErrorString(status) << endl; };
        end=clock();
        printf("\n GPU computations with kernel 1 for %d vectors took %lf s\n\n",limit,1.0*(end-start)/CLOCKS_PER_SEC);
    }
    if (KERNEL == 2) {
        // kernel 2
        start=clock();
        for (i=0;i<Size;i++) {
            for (j=0;j<Size;j++) {
                scalar_2<<< 1, BlockSize, 0>>>(gpuMat , i, j , Len, Size, simMat_gpu );           
            }
        }
        status = hipMemcpy(simMat_cpu, simMat_gpu, sizeof(int)* Size*Size, hipMemcpyDeviceToHost);
        if (status != hipSuccess){ cout << hipGetErrorString(status) << endl; };
        end=clock();
        printf("\n GPU computations with kernel 2 took %lf s\n\n",1.0*(end-start)/CLOCKS_PER_SEC);
        float r = correlation(simMat,simMat_cpu,(Size*Size));
        printf("Correlation between CPU and GPU %f\n",r);
    }

    Min =1.0;
    Max =0.0;
    Sum = 0;

    for (i=0;i<Size;i++) {
        for (j=i+1;j<Size;j++) {
            r = simMat_cpu[i*Size+j];
            r2 = r*r;
            if (r2 < Min) Min=r2;
            if (r2>Max) Max = r2;
            Sum += r2;
        }
    }
    
    printf("Minimum similarity(%%): %f\n",Min*100);
    printf("Maximum similarity(%%): %f\n",Max*100);
    printf("Average similarity(%%): %f\n",Sum/(Size*(Size-1))/2*100);

    for (i=0;i<20;i++)   printf("%d : %8.5f ",i*Size+i,simMat[i*Size+i]);       
    printf("\n");
    for (i=0;i<20;i++)   printf("%d : %8.5f ",i*Size+i,simMat_cpu[i*Size+i]);       
    printf("\n");
    

}


__global__ void scalar_1(float *Mat , int ind_x, int ind_y , int Len, int Size, float *resMat ){
    size_t s = threadIdx.x + blockIdx.x * blockDim.x;
    size_t i;
    int ad_x;
    int ad_y;
    float res;
    // Wektory x i y są fragmentami ciągłego obszaru pamięci w tablicy Mat.
    // Musimy ręcznie policzyć sobie ich adresy i odpowiednie wartości czytać z tablicy Mat.
    
    
    res = 0.0f;
    if (s==0){
        //    *out = 0;
        ad_x = ind_x*Len;
        ad_y = ind_y*Len;
        //printf("i = %d  j = %d ad_x = %d ad_y = %d\n",ind_x,ind_y,ad_x,ad_y);
        //for (i=0; i<Len; i++) {
        for (i=0; i<Len; i++) {
            res+= Mat[ad_x]*Mat[ad_y];
            ad_x++;
            ad_y++;
        }
        // wpisujemy odpowiedni wynik do tablicy wynikowej
        resMat[ind_x*Size+ind_y]=res;
    }
    
}

__global__ void scalar_2(float *Mat, int ind_x, int ind_y, int Len, int Size, float *resMat ){
    size_t s = threadIdx.x + blockIdx.x * blockDim.x;
    int sID = threadIdx.x;
    int i;
    int stride;
    float sum=0;
    int indx;
    int ad_x=ind_x*Len;
    int ad_y=ind_y*Len;
    int loc_x;
    int loc_y;
    
    stride = Len / blockDim.x +1;
    
    
    // We use a single block to process single pair of vectors
    
    __shared__ float pom[BlockSize];
    
    pom[sID] = 0;
    // Najpierw każdy wątek czyta 'stride' wartości z wektorów x i y
    // i wykonuje fragment obliczeń do iloczynu skalarnego
    // Potem wszystkie wątki robią redukcję.
    // Pamiętamy, że każdy wektor jest w rzeczywistości fragmentem pamięci w jednowymiarowej
    // tablicy Mat. Dlatego musimy policzyć odpowiedni adres każdego elementu z tej tablicy
    for (i=0;i<stride;i++) {
        indx = s*stride+i; // Wyliczamy dla każdego wątku numer elementu w wektorach x i y
        loc_x = ad_x + indx; // Wyliczamy numery elementów w tablicy Mat odpowiadające
        loc_y = ad_y + indx; // elementom o numerze indx w wektorach x i y
        if (indx<Len) {
            sum+=Mat[loc_x]*Mat[loc_y];
        }
    }
    pom[sID] = sum;
    __syncthreads(); // synchronizujemy wątki - wszystkie wątki muszą skończyć swoje wczytywanie
    // zanim przejdziemy dalej
    
    // redukcja numer 2
    for (i=1; i<blockDim.x; i*=2){
        if (sID%(2*i)==0){
            pom[sID] += pom[sID + i];
        }
        __syncthreads();
    }
    if (sID==0) {
        // wpisujemy odpowiedni wynik do tablicy wynikowej
        resMat[ind_x*Size+ind_y]=pom[0];
    }

}






#define blockSize 512
#define real float


__global__ void redukcja_1(int N, real* v)
{
 size_t s = threadIdx.x + blockIdx.x * blockDim.x;
 size_t i;

 real p = 0;
 if (s==0){
//	*out = 0;
	for (i=0; i<N; i++)
        //p+= x[i]*y[i];
		p += v[i];
 	v[0] = p;		
 }		
}

__global__ void redukcja_2(int N, real* v, real* out)
{
 size_t s = threadIdx.x + blockIdx.x * blockDim.x;
 int sID = threadIdx.x;
 size_t i;

 __shared__ real pom[blockSize];
 
 pom[sID] = 0;
 if (s<N)
	 pom[sID] = v[s];
 __syncthreads();

 for (i=1; i<blockSize; i*=2){
 	if (sID%(2*i)==0){
		pom[sID] += pom[sID + i];
 	}		
 	__syncthreads();		
 }
 if (sID==0) out[blockIdx.x] = pom[0];
}

float correlation(float *x,float *y,int Size){
    float Sumx, Sumy, Sumxy, Sumxx, Sumyy; 
    float mx, my; 
    float tx, ty;
    float r;

    Sumx =0;
    Sumy =0;
    Sumxy =0;
    Sumxx=0;
    Sumyy=0;

    for (int i=0; i<Size;i++) {
        tx = x[i];
        ty = y[i];
        Sumx += tx;
        Sumy += ty;
        Sumxy += tx*ty;
        Sumxx += tx*tx;
        Sumyy += ty*ty;
    }
    mx = Sumx/Size;
    my = Sumy/Size;
    r = (Sumxx - Size*mx*my)/(sqrt( (Sumxx -Size*mx*mx)*(Sumyy-Size*my*my) ));
    return(r);


}
