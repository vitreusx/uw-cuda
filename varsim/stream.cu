#include "stream.h"
#include "utils.h"

Stream::Stream() {
    check(hipStreamCreate(&stream));
}

Stream::~Stream() {
    check(hipStreamDestroy(stream));
    stream = 0;
}
